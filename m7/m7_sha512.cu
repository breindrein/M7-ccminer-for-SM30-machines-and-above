#include "hip/hip_runtime.h"
/*
 * sha512 djm34
 *
 */

/*
 * sha-512 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_helper.h"
#define SPH_C64(x)    ((uint64_t)(x ## ULL))

//Aus ccminer-msvc2015-0.3.0 von djm kopiert
#define andor64(a,b,c) ((a & (b | c)) | (b & c))
#define xandx64(e,f,g) (g ^ (e & (g ^ f)))
//Ende Kopie

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
//extern int device_sm[8];

__constant__ uint64_t c_PaddedMessage80[16];

static __constant__ uint64_t H_512[8] = {
	SPH_C64(0x6A09E667F3BCC908), SPH_C64(0xBB67AE8584CAA73B),
	SPH_C64(0x3C6EF372FE94F82B), SPH_C64(0xA54FF53A5F1D36F1),
	SPH_C64(0x510E527FADE682D1), SPH_C64(0x9B05688C2B3E6C1F),
	SPH_C64(0x1F83D9ABFB41BD6B), SPH_C64(0x5BE0CD19137E2179)
};

static __constant__ uint64_t const gpu_WK[80] = {
	0x428a2f98d728ae22,0x7137449123ef65cd,0xb5c0fbcfec4d3b2f,0xe9b5dba58189dbbc,
	0x3956c25bf348b538,0x59f111f1b605d019,0x923f82a4af194f9b,0xab1c5ed5da6d8118,
	0xd807aa98a3030242,0x12835b0145706fbe,0x243185be4ee4b28c,0x550c7dc3d5ffb4e2,
	0x72be5d74f27b896f,0x80deb1fe3b1696b1,0x9bdc06a725c71235,0xc19bf174cf692a64,
	0xe49b69c19ef14ad2,0xf0384786384f4472,0xfc19dc68b8cd5b5,0x240ca9dbb7ad9067,
	0x2de92c6f592b0275,0x68f48504aaad3043,0x5cb0a9dcbd41ffa4,0xb3fb89db0bda5464,
	0x99325152ee671cc9,0x7123ae0d31ff6272,0xb0033ff658fdfd45,0xe1fa23485dabbe42,
	0x40e00d5c2dc22ec2,0x560bec2676fa4690,0xf5d468548bee0586,0x7eb60f7f9b748ba,
	0x2b757e0560795790,0xb52d238794343086,0x6adcfb55ae70f872,0x6398611e8a733079,
	0x291459ab496a27f0,0x80c473236c425024,0x232854f0097e8487,0xf30368b14ad940b2,
	0x4505c8f72220c3ac,0x79223106d06018ab,0xa92a9358ef68a70,0x2213ed04d9d16738,
	0xfde1eec528488436,0x1b8e578b3cb41cb3,0xbb29a10ec7f0e115,0x251bea9790505f29,
	0xd4457f79ff355b71,0x2e05d9dbe2066b35,0xb85b71b919d5f399,0xcc59173fcaca449c,
	0x2722a9a858047e9d,0xf102d1898b991e4e,0xf0a86da2960a222b,0x7dce8dba654ef680,
	0x5872e4ced838147c,0x81a93d5212186f4e,0x9b18b478d47f8e8c,0x4d8f8e0e5d851bf6,
	0xcfabc63ef8d4741b,0xa11c143919b22c32,0x743e8af79927636c,0x3d0d0f5b5bc98d7a,
	0x297c0e410c9e6c3c,0x932ee100a01733b3,0xf5b1fd3afc9cd585,0xd8b62f8c1408fe3a,
	0x7192616861de6cf8,0xb4e118d6ac3895b6,0xffd847e31993915c,0x3022c96b9e13ee56,
	0xe00cac4914baa991,0xa8f14913ade567aa,0x2021caba7e45a5a6,0x534bdc6351b491af,
	0x4fdbfb25d8e33212,0xe320be3f9eababf3,0xa3ef683366aff9d0,0xb3ef91a4e5db6e75
};

static __constant__
#if __CUDA_ARCH__ > 500
__align__(16)
#else
__align__(8)
#endif
uint64_t K_512[80] = {
	0x428A2F98D728AE22, 0x7137449123EF65CD, 0xB5C0FBCFEC4D3B2F, 0xE9B5DBA58189DBBC,
	0x3956C25BF348B538, 0x59F111F1B605D019, 0x923F82A4AF194F9B, 0xAB1C5ED5DA6D8118,
	0xD807AA98A3030242, 0x12835B0145706FBE, 0x243185BE4EE4B28C, 0x550C7DC3D5FFB4E2,
	0x72BE5D74F27B896F, 0x80DEB1FE3B1696B1, 0x9BDC06A725C71235, 0xC19BF174CF692694,
	0xE49B69C19EF14AD2, 0xEFBE4786384F25E3, 0x0FC19DC68B8CD5B5, 0x240CA1CC77AC9C65,
	0x2DE92C6F592B0275, 0x4A7484AA6EA6E483, 0x5CB0A9DCBD41FBD4, 0x76F988DA831153B5,
	0x983E5152EE66DFAB, 0xA831C66D2DB43210, 0xB00327C898FB213F, 0xBF597FC7BEEF0EE4,
	0xC6E00BF33DA88FC2, 0xD5A79147930AA725, 0x06CA6351E003826F, 0x142929670A0E6E70,
	0x27B70A8546D22FFC, 0x2E1B21385C26C926, 0x4D2C6DFC5AC42AED, 0x53380D139D95B3DF,
	0x650A73548BAF63DE, 0x766A0ABB3C77B2A8, 0x81C2C92E47EDAEE6, 0x92722C851482353B,
	0xA2BFE8A14CF10364, 0xA81A664BBC423001, 0xC24B8B70D0F89791, 0xC76C51A30654BE30,
	0xD192E819D6EF5218, 0xD69906245565A910, 0xF40E35855771202A, 0x106AA07032BBD1B8,
	0x19A4C116B8D2D0C8, 0x1E376C085141AB53, 0x2748774CDF8EEB99, 0x34B0BCB5E19B48A8,
	0x391C0CB3C5C95A63, 0x4ED8AA4AE3418ACB, 0x5B9CCA4F7763E373, 0x682E6FF3D6B2B8A3,
	0x748F82EE5DEFB2FC, 0x78A5636F43172F60, 0x84C87814A1F0AB72, 0x8CC702081A6439EC,
	0x90BEFFFA23631E28, 0xA4506CEBDE82BDE9, 0xBEF9A3F7B2C67915, 0xC67178F2E372532B,
	0xCA273ECEEA26619C, 0xD186B8C721C0C207, 0xEADA7DD6CDE0EB1E, 0xF57D4F7FEE6ED178,
	0x06F067AA72176FBA, 0x0A637DC5A2C898A6, 0x113F9804BEF90DAE, 0x1B710B35131C471B,
	0x28DB77F523047D84, 0x32CAAB7B40C72493, 0x3C9EBE0A15C9BEBC, 0x431D67C49C100D4C,
	0x4CC5D4BECB3E42B6, 0x597F299CFC657E2A, 0x5FCB6FAB3AD6FAEC, 0x6C44198C4A475817
};


static __device__ __forceinline__ uint64_t bsg5_0(uint64_t x)
{
	uint64_t r1 = ROTR64(x,28);
	uint64_t r2 = ROTR64(x,34);
	uint64_t r3 = ROTR64(x,39);
	return xor3(r1,r2,r3);
}

static __device__ __forceinline__ uint64_t bsg5_1(uint64_t x)
{
	uint64_t r1 = ROTR64(x,14);
	uint64_t r2 = ROTR64(x,18);
	uint64_t r3 = ROTR64(x,41);
	return xor3(r1,r2,r3);
}

static __device__ __forceinline__ uint64_t ssg5_0(uint64_t x)
{
	uint64_t r1 = ROTR64(x,1);
	uint64_t r2 = ROTR64(x,8);
	uint64_t r3 = shr_t64(x,7);
	return xor3(r1,r2,r3);
}

static __device__ __forceinline__ uint64_t ssg5_1(uint64_t x)
{
	uint64_t r1 = ROTR64(x,19);
	uint64_t r2 = ROTR64(x,61);
	uint64_t r3 = shr_t64(x,6);
	return xor3(r1,r2,r3);
}


static __device__ __forceinline__ void sha3_step2(uint64_t* r,uint64_t* W,uint64_t* K,int ord,int i)
{
int u = 8-ord;
uint64_t a=r[(0+u)& 7];
uint64_t b=r[(1+u)& 7];
uint64_t c=r[(2+u)& 7];
uint64_t d=r[(3+u)& 7];
uint64_t e=r[(4+u)& 7];
uint64_t f=r[(5+u)& 7];
uint64_t g=r[(6+u)& 7];
uint64_t h=r[(7+u)& 7];

uint64_t T1, T2;
T1 = h+bsg5_1(e)+xandx64(e,f,g)+W[i]+K[i];
T2 = bsg5_0(a) + andor(a,b,c);
r[(3+u)& 7] = d + T1;
r[(7+u)& 7] = T1 + T2;

}


static __device__ __forceinline__ void sha3_step3(uint64_t* r,const uint64_t* W,int ord,int i)
{
	int u = 8-ord;
	uint64_t a=r[(0+u)& 7];
	uint64_t b=r[(1+u)& 7];
	uint64_t c=r[(2+u)& 7];
	uint64_t d=r[(3+u)& 7];
	uint64_t e=r[(4+u)& 7];
	uint64_t f=r[(5+u)& 7];
	uint64_t g=r[(6+u)& 7];
	uint64_t h=r[(7+u)& 7];

	uint64_t T1, T2;
	T1 = h+bsg5_1(e)+xandx64(e,f,g)+W[i];
	T2 = bsg5_0(a) + andor(a,b,c);
	r[(3+u)& 7] = d + T1;
	r[(7+u)& 7] = T1 + T2;
}


__global__ void sha512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint64_t *inpHash = (uint64_t*)&g_hash + 8*thread;

		uint64_t W[80];
        uint64_t r[8];
#pragma unroll 71
		for (int i=9;i<80;i++) {W[i]=0;}

#pragma unroll 8
 		for (int i = 0; i < 8; i ++) {
			W[i] = cuda_swab64(inpHash[i]);
			r[i] = H_512[i];}

		W[8] = 0x8000000000000000;
		W[15]= 0x0000000000000200;
#pragma unroll 64
		for (int i = 16; i < 80; i ++)
 			W[i] = SPH_T64(ssg5_1(W[i - 2]) + W[i - 7] + ssg5_0(W[i - 15]) + W[i - 16]);

#if __CUDA_ARCH__ < 500    // go figure...
#pragma unroll 10
#endif
		for (int i = 0; i < 10; i ++) {
#pragma unroll 8
			for (int ord=0;ord<8;ord++) {sha3_step2(r,W,K_512,ord,8*i+ord);}
		}

#pragma unroll 8
		for (int i = 0; i < 8; i++) {r[i] = SPH_T64(r[i] + H_512[i]);}

      #pragma unroll 8
      for (int u = 0; u < 8; u ++)
            inpHash[u] = cuda_swab64(r[u]);
	}
}


__global__ void __launch_bounds__(256,3) m7_sha512_gpu50_hash_120(int threads, uint32_t startNounce, uint64_t *outputHash)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
			uint32_t nounce = startNounce + thread;

		uint64_t W[80];
        uint64_t r[8];
#pragma unroll 8
		for (int i = 0; i < 8; i ++) {r[i] = H_512[i];}
#pragma unroll 14
		for (int i = 0; i < 14; i ++) {W[i] = cuda_swab64(c_PaddedMessage80[i]);}
		    W[14] =  cuda_swab64(REPLACE_HIWORD(c_PaddedMessage80[14],nounce));
            W[15] =  cuda_swab64(c_PaddedMessage80[15]);

#pragma unroll 64
		for (int i = 16; i < 80; i ++)
 			W[i] = SPH_T64(ssg5_1(W[i - 2]) + W[i - 7] + ssg5_0(W[i - 15]) + W[i - 16]);

#if __CUDA_ARCH__ < 500    // go figure...
#pragma unroll 10
#endif
		for (int i = 0; i < 10; i ++) {
#pragma unroll 8
			for (int ord=0;ord<8;ord++) {sha3_step2(r,W,K_512,ord,8*i+ord); }
		}
 uint64_t tempr[8];
#pragma unroll 8
		for (int i = 0; i < 8; i++) {tempr[i] = r[i] = SPH_T64(r[i] + H_512[i]);}


#if __CUDA_ARCH__ < 500    // go figure...
#pragma unroll
#endif
		for (int i = 0; i < 10; i ++) {
#pragma unroll 8
			for (int ord=0;ord<8;ord++) {sha3_step3(r,gpu_WK,ord,8*i+ord); }
		}

#pragma unroll 8
for(int i=0;i<8;i++) {outputHash[i*threads+thread] = cuda_swab64(SPH_T64(r[i] + tempr[i]));}

 } /// thread
}


__global__ void __launch_bounds__(256,4) m7_sha512_gpu_hash_120(int threads, uint32_t startNounce, uint64_t *outputHash)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
			uint32_t nounce = startNounce + thread;

		uint64_t W[80];
        uint64_t r[8];
#pragma unroll 8
		for (int i = 0; i < 8; i ++) {r[i] = H_512[i];}
#pragma unroll 14
		for (int i = 0; i < 14; i ++) {W[i] = cuda_swab64(c_PaddedMessage80[i]);}
		    W[14] =  cuda_swab64(REPLACE_HIWORD(c_PaddedMessage80[14],nounce));
            W[15] =  cuda_swab64(c_PaddedMessage80[15]);

#pragma unroll 64
		for (int i = 16; i < 80; i ++)
 			W[i] = SPH_T64(ssg5_1(W[i - 2]) + W[i - 7] + ssg5_0(W[i - 15]) + W[i - 16]);

#if __CUDA_ARCH__ < 500    // go figure...
#pragma unroll 10
#endif
		for (int i = 0; i < 10; i ++) {
#pragma unroll 8
			for (int ord=0;ord<8;ord++) {sha3_step2(r,W,K_512,ord,8*i+ord); }
		}
 uint64_t tempr[8];
#pragma unroll 8
		for (int i = 0; i < 8; i++) {tempr[i] = r[i] = SPH_T64(r[i] + H_512[i]);}


#if __CUDA_ARCH__ < 500    // go figure...
#pragma unroll
#endif
		for (int i = 0; i < 10; i ++) {
#pragma unroll 8
			for (int ord=0;ord<8;ord++) {sha3_step3(r,gpu_WK,ord,8*i+ord); }
		}

#pragma unroll 8
for(int i=0;i<8;i++) {outputHash[i*threads+thread] = cuda_swab64(SPH_T64(r[i] + tempr[i]));}

	} /// thread
}


void m7_sha512_cpu_init(int thr_id, int threads)
{
}


__host__ void sha512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{

	const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	sha512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

	MyStreamSynchronize(NULL, order, thr_id);
}


__host__ void m7_sha512_setBlock_120(void *pdata)
{
	unsigned char PaddedMessage[128];
	uint8_t ending =0x80;
	memcpy(PaddedMessage, pdata, 122);
	memset(PaddedMessage+122,ending,1);
	memset(PaddedMessage+123, 0, 5); //useless
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);

}

__host__ void m7_sha512_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{

	const int threadsperblock = 256;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);
	if (device_sm[thr_id]==5) m7_sha512_gpu50_hash_120<<<grid, block>>>(threads, startNounce, d_outputHash);
	else m7_sha512_gpu_hash_120<<<grid, block>>>(threads, startNounce, d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}

